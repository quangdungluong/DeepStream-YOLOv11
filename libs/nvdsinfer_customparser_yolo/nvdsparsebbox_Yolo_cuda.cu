#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 *
 * Edited by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */

#include <algorithm>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "nvdsinfer_custom_impl.h"

extern "C" bool
NvDsInferParseYoloCuda(std::vector<NvDsInferLayerInfo> const &outputLayersInfo,
                       NvDsInferNetworkInfo const &networkInfo,
                       NvDsInferParseDetectionParams const &detectionParams,
                       std::vector<NvDsInferParseObjectInfo> &objectList);

extern "C" bool
NvDsInferParseYoloECuda(std::vector<NvDsInferLayerInfo> const &outputLayersInfo,
                        NvDsInferNetworkInfo const &networkInfo,
                        NvDsInferParseDetectionParams const &detectionParams,
                        std::vector<NvDsInferParseObjectInfo> &objectList);

__global__ void decodeTensorYoloCuda(NvDsInferParseObjectInfo *binfo,
                                     float *boxes, float *scores,
                                     float *classes, int outputSize, int netW,
                                     int netH, float minPreclusterThreshold) {
  int x_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (x_id >= outputSize) {
    return;
  }

  float maxProb = scores[x_id];
  int maxIndex = (int)classes[x_id];

  if (maxProb < minPreclusterThreshold) {
    binfo[x_id].detectionConfidence = 0.0;
    return;
  }

  float bxc = boxes[x_id * 4 + 0];
  float byc = boxes[x_id * 4 + 1];
  float bw = boxes[x_id * 4 + 2];
  float bh = boxes[x_id * 4 + 3];

  float x0 = bxc - bw / 2;
  float y0 = byc - bh / 2;
  float x1 = x0 + bw;
  float y1 = y0 + bh;

  x0 = fminf(float(netW), fmaxf(float(0.0), x0));
  y0 = fminf(float(netH), fmaxf(float(0.0), y0));
  x1 = fminf(float(netW), fmaxf(float(0.0), x1));
  y1 = fminf(float(netH), fmaxf(float(0.0), y1));

  binfo[x_id].left = x0;
  binfo[x_id].top = y0;
  binfo[x_id].width = fminf(float(netW), fmaxf(float(0.0), x1 - x0));
  binfo[x_id].height = fminf(float(netH), fmaxf(float(0.0), y1 - y0));
  binfo[x_id].detectionConfidence = maxProb;
  binfo[x_id].classId = maxIndex;
}

__global__ void decodeTensorYoloECuda(NvDsInferParseObjectInfo *binfo,
                                      float *boxes, float *scores,
                                      float *classes, int outputSize, int netW,
                                      int netH, float minPreclusterThreshold) {
  int x_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (x_id >= outputSize) {
    return;
  }

  float maxProb = scores[x_id];
  int maxIndex = (int)classes[x_id];

  if (maxProb < minPreclusterThreshold) {
    binfo[x_id].detectionConfidence = 0.0;
    return;
  }

  float x0 = boxes[x_id * 4 + 0];
  float y0 = boxes[x_id * 4 + 1];
  float x1 = boxes[x_id * 4 + 2];
  float y1 = boxes[x_id * 4 + 3];

  x0 = fminf(float(netW), fmaxf(float(0.0), x0));
  y0 = fminf(float(netH), fmaxf(float(0.0), y0));
  x1 = fminf(float(netW), fmaxf(float(0.0), x1));
  y1 = fminf(float(netH), fmaxf(float(0.0), y1));

  binfo[x_id].left = x0;
  binfo[x_id].top = y0;
  binfo[x_id].width = fminf(float(netW), fmaxf(float(0.0), x1 - x0));
  binfo[x_id].height = fminf(float(netH), fmaxf(float(0.0), y1 - y0));
  binfo[x_id].detectionConfidence = maxProb;
  binfo[x_id].classId = maxIndex;
}

static bool NvDsInferParseCustomYoloCuda(
    std::vector<NvDsInferLayerInfo> const &outputLayersInfo,
    NvDsInferNetworkInfo const &networkInfo,
    NvDsInferParseDetectionParams const &detectionParams,
    std::vector<NvDsInferParseObjectInfo> &objectList) {
  if (outputLayersInfo.empty()) {
    std::cerr << "ERROR: Could not find output layer in bbox parsing"
              << std::endl;
    return false;
  }

  const NvDsInferLayerInfo &boxes = outputLayersInfo[0];
  const NvDsInferLayerInfo &scores = outputLayersInfo[1];
  const NvDsInferLayerInfo &classes = outputLayersInfo[2];

  const int outputSize = boxes.inferDims.d[0];

  thrust::device_vector<NvDsInferParseObjectInfo> objects(outputSize);

  float minPreclusterThreshold =
      *(std::min_element(detectionParams.perClassPreclusterThreshold.begin(),
                         detectionParams.perClassPreclusterThreshold.end()));

  int threads_per_block = 1024;
  int number_of_blocks = ((outputSize - 1) / threads_per_block) + 1;

  decodeTensorYoloCuda<<<number_of_blocks, threads_per_block>>>(
      thrust::raw_pointer_cast(objects.data()), (float *)(boxes.buffer),
      (float *)(scores.buffer), (float *)(classes.buffer), outputSize,
      networkInfo.width, networkInfo.height, minPreclusterThreshold);

  objectList.resize(outputSize);
  thrust::copy(objects.begin(), objects.end(), objectList.begin());

  return true;
}

static bool NvDsInferParseCustomYoloECuda(
    std::vector<NvDsInferLayerInfo> const &outputLayersInfo,
    NvDsInferNetworkInfo const &networkInfo,
    NvDsInferParseDetectionParams const &detectionParams,
    std::vector<NvDsInferParseObjectInfo> &objectList) {
  if (outputLayersInfo.empty()) {
    std::cerr << "ERROR: Could not find output layer in bbox parsing"
              << std::endl;
    return false;
  }

  const NvDsInferLayerInfo &boxes = outputLayersInfo[0];
  const NvDsInferLayerInfo &scores = outputLayersInfo[1];
  const NvDsInferLayerInfo &classes = outputLayersInfo[2];

  const int outputSize = boxes.inferDims.d[0];

  thrust::device_vector<NvDsInferParseObjectInfo> objects(outputSize);

  float minPreclusterThreshold =
      *(std::min_element(detectionParams.perClassPreclusterThreshold.begin(),
                         detectionParams.perClassPreclusterThreshold.end()));

  int threads_per_block = 1024;
  int number_of_blocks = ((outputSize - 1) / threads_per_block) + 1;

  decodeTensorYoloECuda<<<number_of_blocks, threads_per_block>>>(
      thrust::raw_pointer_cast(objects.data()), (float *)(boxes.buffer),
      (float *)(scores.buffer), (float *)(classes.buffer), outputSize,
      networkInfo.width, networkInfo.height, minPreclusterThreshold);

  objectList.resize(outputSize);
  thrust::copy(objects.begin(), objects.end(), objectList.begin());

  return true;
}

extern "C" bool
NvDsInferParseYoloCuda(std::vector<NvDsInferLayerInfo> const &outputLayersInfo,
                       NvDsInferNetworkInfo const &networkInfo,
                       NvDsInferParseDetectionParams const &detectionParams,
                       std::vector<NvDsInferParseObjectInfo> &objectList) {
  return NvDsInferParseCustomYoloCuda(outputLayersInfo, networkInfo,
                                      detectionParams, objectList);
}

extern "C" bool
NvDsInferParseYoloECuda(std::vector<NvDsInferLayerInfo> const &outputLayersInfo,
                        NvDsInferNetworkInfo const &networkInfo,
                        NvDsInferParseDetectionParams const &detectionParams,
                        std::vector<NvDsInferParseObjectInfo> &objectList) {
  return NvDsInferParseCustomYoloECuda(outputLayersInfo, networkInfo,
                                       detectionParams, objectList);
}

CHECK_CUSTOM_PARSE_FUNC_PROTOTYPE(NvDsInferParseYoloCuda);
CHECK_CUSTOM_PARSE_FUNC_PROTOTYPE(NvDsInferParseYoloECuda);
