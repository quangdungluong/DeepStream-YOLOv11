/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void
gpuYoloLayer_nc(const float *input, float *boxes, float *scores, float *classes,
                const uint netWidth, const uint netHeight, const uint gridSizeX,
                const uint gridSizeY, const uint numOutputClasses,
                const uint numBBoxes, const uint64_t lastInputSize,
                const float scaleXY, const float *anchors, const int *mask) {
  uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
  uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
  uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

  if (x_id >= gridSizeX || y_id >= gridSizeY || z_id >= numBBoxes)
    return;

  const int numGridCells = gridSizeX * gridSizeY;
  const int bbindex = y_id * gridSizeX + x_id;

  const float alpha = scaleXY;
  const float beta = -0.5 * (scaleXY - 1);

  float xc =
      (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)] *
           alpha +
       beta + x_id) *
      netWidth / gridSizeX;

  float yc =
      (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)] *
           alpha +
       beta + y_id) *
      netHeight / gridSizeY;

  float w =
      __powf(
          input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)] *
              2,
          2) *
      anchors[mask[z_id] * 2];

  float h =
      __powf(
          input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)] *
              2,
          2) *
      anchors[mask[z_id] * 2 + 1];

  const float objectness =
      input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)];

  float maxProb = 0.0f;
  int maxIndex = -1;

  for (uint i = 0; i < numOutputClasses; ++i) {
    float prob = input[bbindex + numGridCells *
                                     (z_id * (5 + numOutputClasses) + (5 + i))];
    if (prob > maxProb) {
      maxProb = prob;
      maxIndex = i;
    }
  }

  int count =
      z_id * gridSizeX * gridSizeY + y_id * gridSizeY + x_id + lastInputSize;

  boxes[count * 4 + 0] = xc;
  boxes[count * 4 + 1] = yc;
  boxes[count * 4 + 2] = w;
  boxes[count * 4 + 3] = h;
  scores[count] = maxProb * objectness;
  classes[count] = (float)maxIndex;
}

hipError_t
cudaYoloLayer_nc(const void *input, void *boxes, void *scores, void *classes,
                 const uint &batchSize, const uint64_t &inputSize,
                 const uint64_t &outputSize, const uint64_t &lastInputSize,
                 const uint &netWidth, const uint &netHeight,
                 const uint &gridSizeX, const uint &gridSizeY,
                 const uint &numOutputClasses, const uint &numBBoxes,
                 const float &scaleXY, const void *anchors, const void *mask,
                 hipStream_t stream);

hipError_t
cudaYoloLayer_nc(const void *input, void *boxes, void *scores, void *classes,
                 const uint &batchSize, const uint64_t &inputSize,
                 const uint64_t &outputSize, const uint64_t &lastInputSize,
                 const uint &netWidth, const uint &netHeight,
                 const uint &gridSizeX, const uint &gridSizeY,
                 const uint &numOutputClasses, const uint &numBBoxes,
                 const float &scaleXY, const void *anchors, const void *mask,
                 hipStream_t stream) {
  dim3 threads_per_block(16, 16, 4);
  dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1,
                        (gridSizeY / threads_per_block.y) + 1,
                        (numBBoxes / threads_per_block.z) + 1);

  for (unsigned int batch = 0; batch < batchSize; ++batch) {
    gpuYoloLayer_nc<<<number_of_blocks, threads_per_block, 0, stream>>>(
        reinterpret_cast<const float *>(input) + (batch * inputSize),
        reinterpret_cast<float *>(boxes) + (batch * 4 * outputSize),
        reinterpret_cast<float *>(scores) + (batch * 1 * outputSize),
        reinterpret_cast<float *>(classes) + (batch * 1 * outputSize), netWidth,
        netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes,
        lastInputSize, scaleXY, reinterpret_cast<const float *>(anchors),
        reinterpret_cast<const int *>(mask));
  }
  return hipGetLastError();
}
