/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

inline __device__ float sigmoidGPU(const float &x) {
  return 1.0f / (1.0f + __expf(-x));
}

__global__ void gpuYoloLayer(const float *input, float *boxes, float *scores,
                             float *classes, const uint netWidth,
                             const uint netHeight, const uint gridSizeX,
                             const uint gridSizeY, const uint numOutputClasses,
                             const uint numBBoxes, const uint64_t lastInputSize,
                             const float scaleXY, const float *anchors,
                             const int *mask) {
  uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
  uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
  uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

  if (x_id >= gridSizeX || y_id >= gridSizeY || z_id >= numBBoxes)
    return;

  const int numGridCells = gridSizeX * gridSizeY;
  const int bbindex = y_id * gridSizeX + x_id;

  const float alpha = scaleXY;
  const float beta = -0.5 * (scaleXY - 1);

  float xc =
      (sigmoidGPU(input[bbindex +
                        numGridCells * (z_id * (5 + numOutputClasses) + 0)]) *
           alpha +
       beta + x_id) *
      netWidth / gridSizeX;

  float yc =
      (sigmoidGPU(input[bbindex +
                        numGridCells * (z_id * (5 + numOutputClasses) + 1)]) *
           alpha +
       beta + y_id) *
      netHeight / gridSizeY;

  float w =
      __expf(
          input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]) *
      anchors[mask[z_id] * 2];

  float h =
      __expf(
          input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]) *
      anchors[mask[z_id] * 2 + 1];

  const float objectness = sigmoidGPU(
      input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

  float maxProb = 0.0f;
  int maxIndex = -1;

  for (uint i = 0; i < numOutputClasses; ++i) {
    float prob = sigmoidGPU(
        input[bbindex +
              numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))]);
    if (prob > maxProb) {
      maxProb = prob;
      maxIndex = i;
    }
  }

  int count =
      z_id * gridSizeX * gridSizeY + y_id * gridSizeY + x_id + lastInputSize;

  boxes[count * 4 + 0] = xc;
  boxes[count * 4 + 1] = yc;
  boxes[count * 4 + 2] = w;
  boxes[count * 4 + 3] = h;
  scores[count] = maxProb * objectness;
  classes[count] = (float)maxIndex;
}

hipError_t cudaYoloLayer(const void *input, void *boxes, void *scores,
                          void *classes, const uint &batchSize,
                          const uint64_t &inputSize, const uint64_t &outputSize,
                          const uint64_t &lastInputSize, const uint &netWidth,
                          const uint &netHeight, const uint &gridSizeX,
                          const uint &gridSizeY, const uint &numOutputClasses,
                          const uint &numBBoxes, const float &scaleXY,
                          const void *anchors, const void *mask,
                          hipStream_t stream);

hipError_t cudaYoloLayer(const void *input, void *boxes, void *scores,
                          void *classes, const uint &batchSize,
                          const uint64_t &inputSize, const uint64_t &outputSize,
                          const uint64_t &lastInputSize, const uint &netWidth,
                          const uint &netHeight, const uint &gridSizeX,
                          const uint &gridSizeY, const uint &numOutputClasses,
                          const uint &numBBoxes, const float &scaleXY,
                          const void *anchors, const void *mask,
                          hipStream_t stream) {
  dim3 threads_per_block(16, 16, 4);
  dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1,
                        (gridSizeY / threads_per_block.y) + 1,
                        (numBBoxes / threads_per_block.z) + 1);

  for (unsigned int batch = 0; batch < batchSize; ++batch) {
    gpuYoloLayer<<<number_of_blocks, threads_per_block, 0, stream>>>(
        reinterpret_cast<const float *>(input) + (batch * inputSize),
        reinterpret_cast<float *>(boxes) + (batch * 4 * outputSize),
        reinterpret_cast<float *>(scores) + (batch * 1 * outputSize),
        reinterpret_cast<float *>(classes) + (batch * 1 * outputSize), netWidth,
        netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes,
        lastInputSize, scaleXY, reinterpret_cast<const float *>(anchors),
        reinterpret_cast<const int *>(mask));
  }
  return hipGetLastError();
}
