/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

inline __device__ float sigmoidGPU(const float &x) {
  return 1.0f / (1.0f + __expf(-x));
}

__device__ void softmaxGPU(const float *input, const int bbindex,
                           const int numGridCells, uint z_id,
                           const uint numOutputClasses, float temp,
                           float *output) {
  int i;
  float sum = 0;
  float largest = -INFINITY;
  for (i = 0; i < numOutputClasses; ++i) {
    int val = input[bbindex +
                    numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];
    largest = (val > largest) ? val : largest;
  }
  for (i = 0; i < numOutputClasses; ++i) {
    float e =
        __expf(input[bbindex +
                     numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] /
                   temp -
               largest / temp);
    sum += e;
    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] =
        e;
  }
  for (i = 0; i < numOutputClasses; ++i) {
    output[bbindex +
           numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] /= sum;
  }
}

__global__ void
gpuRegionLayer(const float *input, float *softmax, float *boxes, float *scores,
               float *classes, const uint netWidth, const uint netHeight,
               const uint gridSizeX, const uint gridSizeY,
               const uint numOutputClasses, const uint numBBoxes,
               const uint64_t lastInputSize, const float *anchors) {
  uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
  uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
  uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

  if (x_id >= gridSizeX || y_id >= gridSizeY || z_id >= numBBoxes)
    return;

  const int numGridCells = gridSizeX * gridSizeY;
  const int bbindex = y_id * gridSizeX + x_id;

  float xc =
      (sigmoidGPU(input[bbindex +
                        numGridCells * (z_id * (5 + numOutputClasses) + 0)]) +
       x_id) *
      netWidth / gridSizeX;

  float yc =
      (sigmoidGPU(input[bbindex +
                        numGridCells * (z_id * (5 + numOutputClasses) + 1)]) +
       y_id) *
      netHeight / gridSizeY;

  float w =
      __expf(
          input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]) *
      anchors[z_id * 2] * netWidth / gridSizeX;

  float h =
      __expf(
          input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]) *
      anchors[z_id * 2 + 1] * netHeight / gridSizeY;

  const float objectness = sigmoidGPU(
      input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

  softmaxGPU(input, bbindex, numGridCells, z_id, numOutputClasses, 1.0,
             softmax);

  float maxProb = 0.0f;
  int maxIndex = -1;

  for (uint i = 0; i < numOutputClasses; ++i) {
    float prob =
        softmax[bbindex +
                numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];
    if (prob > maxProb) {
      maxProb = prob;
      maxIndex = i;
    }
  }

  int count =
      z_id * gridSizeX * gridSizeY + y_id * gridSizeY + x_id + lastInputSize;

  boxes[count * 4 + 0] = xc;
  boxes[count * 4 + 1] = yc;
  boxes[count * 4 + 2] = w;
  boxes[count * 4 + 3] = h;
  scores[count] = maxProb * objectness;
  classes[count] = (float)maxIndex;
}

hipError_t cudaRegionLayer(const void *input, void *softmax, void *boxes,
                            void *scores, void *classes, const uint &batchSize,
                            const uint64_t &inputSize,
                            const uint64_t &outputSize,
                            const uint64_t &lastInputSize, const uint &netWidth,
                            const uint &netHeight, const uint &gridSizeX,
                            const uint &gridSizeY, const uint &numOutputClasses,
                            const uint &numBBoxes, const void *anchors,
                            hipStream_t stream);

hipError_t cudaRegionLayer(const void *input, void *softmax, void *boxes,
                            void *scores, void *classes, const uint &batchSize,
                            const uint64_t &inputSize,
                            const uint64_t &outputSize,
                            const uint64_t &lastInputSize, const uint &netWidth,
                            const uint &netHeight, const uint &gridSizeX,
                            const uint &gridSizeY, const uint &numOutputClasses,
                            const uint &numBBoxes, const void *anchors,
                            hipStream_t stream) {
  dim3 threads_per_block(16, 16, 4);
  dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1,
                        (gridSizeY / threads_per_block.y) + 1,
                        (numBBoxes / threads_per_block.z) + 1);

  for (unsigned int batch = 0; batch < batchSize; ++batch) {
    gpuRegionLayer<<<number_of_blocks, threads_per_block, 0, stream>>>(
        reinterpret_cast<const float *>(input) + (batch * inputSize),
        reinterpret_cast<float *>(softmax) + (batch * inputSize),
        reinterpret_cast<float *>(boxes) + (batch * 4 * outputSize),
        reinterpret_cast<float *>(scores) + (batch * 1 * outputSize),
        reinterpret_cast<float *>(classes) + (batch * 1 * outputSize), netWidth,
        netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes,
        lastInputSize, reinterpret_cast<const float *>(anchors));
  }
  return hipGetLastError();
}
